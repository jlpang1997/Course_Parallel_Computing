
#include "hip/hip_runtime.h"

#include<iostream>
#include <stdio.h>
#define CUDACC
using namespace std;
// 矩阵类型，行优先，M(row, col) = *(M.elements + row * M.width + col)
struct Matrix
{
	int width;
	int height;
	int *elements;
};
// 获取矩阵A的(row, col)元素
__device__ int getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}
__device__ void print(int out)
{
	//printf("%d");
}

// 矩阵相加kernel，2-D，每个线程计算一个元素
__global__ void matAddKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	Cvalue = getElement(A, row, col) + getElement(B, row, col);
	setElement(C, row, col, Cvalue);
}
// 矩阵相乘kernel，2-D，每个线程计算一个元素
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;//获取该线程所处理的矩阵行号
	int col = threadIdx.x + blockIdx.x * blockDim.x;//获取该线程所处理的矩阵列号

	for (int i = 0; i < A->width; ++i)//普通的矩阵乘法
	{
		int a, b, c;
		a = A->elements[row*A->width + i];
		b = B->elements[i*B->width + col];
		c = a * b;
		Cvalue +=c;
	}
	setElement(C, row, col, Cvalue);//把结果写回到C矩阵
}
//using namespace std;
int main()
{
	int width = 1 << 2;
	int height = 1 << 2;
	Matrix *A, *B, *C, *D;
	

	int height_A = 16;
	int width_A = 4;
	int height_B = width_A;
	int width_B = 4;

	int width_result = width_B;
	int height_result = height_A;

	// 申请托管内存  不用ppt上的拷来拷去的做法 这相当于是共享内存了吧
	hipMallocManaged((void**)&A, sizeof(Matrix));
	hipMallocManaged((void**)&B, sizeof(Matrix));
	hipMallocManaged((void**)&C, sizeof(Matrix));
	hipMallocManaged((void**)&D, sizeof(Matrix));

	hipMallocManaged((void**)&A->elements, height_A*width_A*sizeof(int));
	hipMallocManaged((void**)&B->elements, height_B*width_B * sizeof(int));
	hipMallocManaged((void**)&C->elements, height_result*width_result * sizeof(int));
	hipMallocManaged((void**)&D->elements, height_result*width_result * sizeof(int));

	A->height = height_A;
	A->width = width_A;
	B->height = height_B;
	B->width = width_B;
	C->height = height_result;
	C->width = width_result;
	D->height = height_result;
	D->width = width_result;

	for (int i = 0; i < height_A; i++)
	{
		for (int j = 0; j < width_A; j++)
		{
			A->elements[i*width_A + j]=rand()%10;
		}
	}
	for (int i = 0; i < height_B; i++)
	{
		for (int j = 0; j < width_B; j++)
		{
			B->elements[i*width_B + j]=rand()%10;
		}
	}
	dim3 blockSize(2,2);
	dim3 gridSize(width_B/blockSize.x,height_A/blockSize.y);
	// 执行kernel
	matMulKernel <<< gridSize, blockSize >>> (A, B, C);
	 //同步device 保证结果能正确访问

	hipDeviceSynchronize();
	// 检查执行结果


	std::cout << "A:" << std::endl;
	for (int i = 0; i < height_A; ++i)
	{
		for (int j = 0; j < width_A; j++)
		{
			printf("%3d ", A->elements[i*width_A + j]);
		}
		std::cout << std::endl;
	}
	std::cout << "B:" << std::endl;
	for (int i = 0; i < height_B; ++i)
	{
		for (int j = 0; j < width_B; j++)
		{
			printf("%3d ", B->elements[i*width_B + j]);
		}
		std::cout << std::endl;
	}
	std::cout << "\n\n\nA*B:" << std::endl;
	for (int i = 0; i < height_result; i++)
	{
		for (int j = 0; j < width_result; j++)
		{
			printf("%10d ", C->elements[i*width_result + j]);
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;

	return 0;
}

/*
	1,学会用Nsight调试
	2，学会一些基本的cuda函数
	3，完成实验2



*/
