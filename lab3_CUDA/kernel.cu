
#include "hip/hip_runtime.h"

#include<iostream>
#include <stdio.h>
#define CUDACC
using namespace std;
// �������ͣ������ȣ�M(row, col) = *(M.elements + row * M.width + col)
struct Matrix
{
	int width;
	int height;
	int *elements;
};
// ��ȡ����A��(row, col)Ԫ��
__device__ int getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// Ϊ����A��(row, col)Ԫ�ظ�ֵ
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}
__device__ void print(int out)
{
	//printf("%d");
}

// �������kernel��2-D��ÿ���̼߳���һ��Ԫ��
__global__ void matAddKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	Cvalue = getElement(A, row, col) + getElement(B, row, col);
	setElement(C, row, col, Cvalue);
}
// �������kernel��2-D��ÿ���̼߳���һ��Ԫ��
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{

	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;//��ȡ���߳�������ľ����к�
	int col = threadIdx.x + blockIdx.x * blockDim.x;//��ȡ���߳�������ľ����к�

	//col = 0;
	//printf("(%d , %d)\n", row, col);
	for (int i = 0; i < A->height; i++)
	{
		for (int j = 0; j < A->width; j++)
		{
			printf("%d ", A->elements[i*A->width + j]);
		}
		printf("\n");
	}
	for (int i = 0; i < B->height; i++)
	{
		for (int j = 0; j < B->width; j++)
		{
			printf("%d ", B->elements[i*A->width + j]);
		}
		printf("\n");
	}
	for (int i = 0; i < A->width; ++i)//��ͨ�ľ���˷�
	{
		int a, b, c;
		//a = getElement(A, row, i);
		a = A->elements[row*A->width + i];
		b = B->elements[i*B->width + col];
		//b = getElement(B, i, col);
		c = a * b;
		Cvalue +=c;
		printf("%d * %d = %d \n", a,b,c);
	}
	//printf("\n");
	setElement(C, row, col, Cvalue);//�ѽ��д�ص�C����
}
//using namespace std;
int main()
{
	int width = 1 << 2;
	int height = 1 << 2;
	Matrix *A, *B, *C, *D;
	// �����й��ڴ�

	int width_A = 4;
	int height_A = 4;
	int height_B = width_A;
	int width_B = 1;

	int width_result = width_B;
	int height_result = height_A;


	hipMallocManaged((void**)&A, sizeof(Matrix));
	hipMallocManaged((void**)&B, sizeof(Matrix));
	hipMallocManaged((void**)&C, sizeof(Matrix));
	hipMallocManaged((void**)&D, sizeof(Matrix));

	hipMallocManaged((void**)&A->elements, height_A*width_A*sizeof(int));
	hipMallocManaged((void**)&B->elements, height_B*width_B * sizeof(int));
	hipMallocManaged((void**)&C->elements, height_result*width_result * sizeof(int));
	hipMallocManaged((void**)&D->elements, height_result*width_result * sizeof(int));

	A->height = height_A;
	A->width = width_A;
	B->height = height_B;
	B->width = width_B;
	C->height = height_result;
	C->width = width_result;
	D->height = height_result;
	D->width = width_result;

	for (int i = 0; i < height_A; i++)
	{
		for (int j = 0; j < width_A; j++)
		{
			A->elements[i*height_A + j]=rand()%10;
		}
	}
	for (int i = 0; i < height_B; i++)
	{
		for (int j = 0; j < width_B; j++)
		{
			B->elements[i*height_B + j]=rand()%10;
		}
	}

	//dim3 blockSize(1, 2);
	//dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
	//	(height + blockSize.y - 1) / blockSize.y);
	////std::cout << (width + blockSize.x - 1) / blockSize.x << std::endl;
	//// ִ��kernel
	//matMulKernel <<< gridSize, blockSize >>> (A, B, C);
	 //ͬ��device ��֤�������ȷ����


	dim3 blockSize(1, 2);
	dim3 gridSize((width_result + blockSize.x - 1) / blockSize.x,
		(height_result + blockSize.y - 1) / blockSize.y);
	// ִ��kernel
	matMulKernel <<< gridSize, blockSize >>> (A, B, C);
	 //ͬ��device ��֤�������ȷ����

	hipDeviceSynchronize();
	// ���ִ�н��


	std::cout << "A:" << std::endl;
	for (int i = 0; i < height_A; ++i)
	{
		for (int j = 0; j < width_A; j++)
		{
			printf("%3d ", A->elements[i*height_A + j]);
		}
		std::cout << std::endl;
	}
	std::cout << "B:" << std::endl;
	for (int i = 0; i < height_B; ++i)
	{
		for (int j = 0; j < width_B; j++)
		{
			printf("%3d ", B->elements[i*height_B + j]);
		}
		std::cout << std::endl;
	}
	std::cout << "A*B:" << std::endl;
	for (int i = 0; i < height_result; ++i)
	{
		for (int j = 0; j < width_result; j++)
		{
			printf("%3d ", C->elements[i*height_result + j]);
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;

	return 0;
}
