#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <stdio.h>
#define CUDACC


void MatrixMulti_ShareMem(int *A, int *B, int *C, int n);
__global__ void MatrixMulti_ShareMem_device(int *A, int *B, int *C, int n);
using namespace std;
// �������ͣ������ȣ�M(row, col) = *(M.elements + row * M.width + col)
struct Matrix
{
	int width;
	int height;
	int *elements;
};
// ��ȡ����A��(row, col)Ԫ��
__device__ int getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// Ϊ����A��(row, col)Ԫ�ظ�ֵ
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}
__device__ void print(int out)
{
	//printf("%d");
}

// �������kernel��2-D��ÿ���̼߳���һ��Ԫ��
__global__ void matAddKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	Cvalue = getElement(A, row, col) + getElement(B, row, col);
	setElement(C, row, col, Cvalue);
}
// �������kernel��2-D��ÿ���̼߳���һ��Ԫ��
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;//��ȡ���߳�������ľ����к�
	int col = threadIdx.x + blockIdx.x * blockDim.x;//��ȡ���߳�������ľ����к�

	for (int i = 0; i < A->width; ++i)//��ͨ�ľ���˷�
	{
		int a, b, c;
		a = A->elements[row*A->width + i];
		b = B->elements[i*B->width + col];
		c = a * b;
		Cvalue +=c;
	}
	setElement(C, row, col, Cvalue);//�ѽ��д�ص�C����
}
//using namespace std;
int main()
{
	//int width = 1 << 2;
	//int height = 1 << 2;
	//Matrix *A, *B, *C, *D;
	

	int height_A = 4;
	int width_A = 4;
	int height_B = width_A;
	int width_B = 4;

	int width_result = width_B;
	int height_result = height_A;

	//// �����й��ڴ�  ����ppt�ϵĿ�����ȥ������ ���൱���ǹ����ڴ��˰�
	//hipMallocManaged((void**)&A, sizeof(Matrix));
	//hipMallocManaged((void**)&B, sizeof(Matrix));
	//hipMallocManaged((void**)&C, sizeof(Matrix));
	//hipMallocManaged((void**)&D, sizeof(Matrix));

	//hipMallocManaged((void**)&A->elements, height_A*width_A*sizeof(int));
	//hipMallocManaged((void**)&B->elements, height_B*width_B * sizeof(int));
	//hipMallocManaged((void**)&C->elements, height_result*width_result * sizeof(int));
	//hipMallocManaged((void**)&D->elements, height_result*width_result * sizeof(int));

	//A->height = height_A;
	//A->width = width_A;
	//B->height = height_B;
	//B->width = width_B;
	//C->height = height_result;
	//C->width = width_result;
	//D->height = height_result;
	//D->width = width_result;

	//for (int i = 0; i < height_A; i++)
	//{
	//	for (int j = 0; j < width_A; j++)
	//	{
	//		A->elements[i*width_A + j]=rand()%10;
	//	}
	//}
	//for (int i = 0; i < height_B; i++)
	//{
	//	for (int j = 0; j < width_B; j++)
	//	{
	//		B->elements[i*width_B + j]=rand()%10;
	//	}
	//}
	//dim3 blockSize(1,2);
	//dim3 gridSize(width_B/blockSize.x,height_A/blockSize.y);
	//// ִ��kernel
	//matMulKernel <<< gridSize, blockSize >>> (A, B, C);
	// //ͬ��device ��֤�������ȷ����

	//hipDeviceSynchronize();
	//// ���ִ�н��


	//std::cout << "A:" << std::endl;
	//for (int i = 0; i < height_A; ++i)
	//{
	//	for (int j = 0; j < width_A; j++)
	//	{
	//		printf("%3d ", A->elements[i*width_A + j]);
	//	}
	//	std::cout << std::endl;
	//}
	//std::cout << "B:" << std::endl;
	//for (int i = 0; i < height_B; ++i)
	//{
	//	for (int j = 0; j < width_B; j++)
	//	{
	//		printf("%3d ", B->elements[i*width_B + j]);
	//	}
	//	std::cout << std::endl;
	//}
	//std::cout << "\n\n\nA*B:" << std::endl;
	//for (int i = 0; i < height_result; i++)
	//{
	//	for (int j = 0; j < width_result; j++)
	//	{
	//		printf("%10d ", C->elements[i*width_result + j]);
	//	}
	//	std::cout << std::endl;
	//}
	//std::cout << std::endl;


	int *A = (int *)malloc(sizeof(int)*height_A*width_A);
	int *B = (int*)malloc(sizeof(int)*height_B*width_B);
	int*C = (int *)malloc(sizeof(int)*height_result*width_result);
	for (int i = 0; i < height_A; i++)
	{
		for (int j = 0; j < width_A; j++)
		{
			A[i*width_A + j]=rand()%10;
			printf("%3d ", A[i*width_A + j]);
		}
		printf("\n");
	}
	cout << endl;

	for (int i = 0; i < height_B; i++)
	{
		for (int j = 0; j < width_B; j++)
		{
			B[i*width_B + j]=rand()%10;
			printf("%3d ", B[i*width_B + j]);
		}
		printf("\n");
	}
	cout << endl << endl;
	MatrixMulti_ShareMem(A, B, C, height_A);
	for (int i = 0; i < height_result; i++)
	{
		for (int j = 0; j < width_result; j++)
		{
			printf("%3d ", C[i*width_result + j]);
		}
		printf("\n");
	}
	free(A);
	free(B);
	free(C);
	return 0;
}
void MatrixMulti_ShareMem(int *A, int *B, int *C, int n)
{
	int *cuda_A, *cuda_B, *cuda_C;
	int size = sizeof(int)*n*n;
	hipMalloc(&cuda_A, size);
	hipMalloc(&cuda_B, size);
	hipMalloc(&cuda_C, size);//cuda��ȫ���ڴ�
	
	hipMemcpy(cuda_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_B, B, size, hipMemcpyHostToDevice);
	//����������ŵ�gpu��ȫ���ڴ���

	dim3 blocksize(2, 2);
	dim3 gridsize(n / blocksize.x, n / blocksize.y);



	MatrixMulti_ShareMem_device << <gridsize, blocksize ,blocksize.x*blocksize.y*sizeof(int)>> > (cuda_A, cuda_B, cuda_C, n);//Ҫָ����̬����Ĺ����ڴ��С

	hipMemcpy(C, cuda_C,size, hipMemcpyDeviceToHost);
	hipFree(cuda_A);
	hipFree(cuda_B);
	hipFree(cuda_C);
}


__global__ void MatrixMulti_ShareMem_device(int *A, int *B, int *C, int n)
{
	int x_start_A = blockIdx.x*blockDim.x;
	int y_start_A = blockIdx.y*blockDim.y;
	
	//���߳�����block�ĵ�һ��Ԫ�ص�����
	int x_start_B= blockIdx.x*blockDim.x;
	int y_start_B= blockIdx.y*blockDim.y;


	int value = 0;
	for (int i = 0; i < n / blockDim.x; i++)
	{
		x_start_A += i * blockDim.x;
		y_start_B += i * blockDim.y;

		extern __shared__  int matrix_A[],matrix_B[];

		matrix_A[threadIdx.y*blockDim.x + threadIdx.x] = A[(y_start_A+threadIdx.y)*n + x_start_A+threadIdx.x];

		matrix_B[threadIdx.y*blockDim.x + threadIdx.y] = B[(y_start_B + threadIdx.y)*n + x_start_B + threadIdx.x];

		//__syncthreads();

		for (int i = 0; i < blockDim.x; i++)
		{
			value += matrix_A[threadIdx.y*blockDim.x + i] * matrix_B[i*blockDim.x + threadIdx.x];
		}

	}


}


/*
	1,ѧ����Nsight����
	2��ѧ��һЩ������cuda����
	3�����ʵ��2



*/
