
#include "hip/hip_runtime.h"

#include<iostream>
#include <stdio.h>
#define CUDACC
using namespace std;
// �������ͣ������ȣ�M(row, col) = *(M.elements + row * M.width + col)
struct Matrix
{
	int width;
	int height;
	int *elements;
};
// ��ȡ����A��(row, col)Ԫ��
__device__ int getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// Ϊ����A��(row, col)Ԫ�ظ�ֵ
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}
__device__ void print(int out)
{
	//printf("%d");
}

// �������kernel��2-D��ÿ���̼߳���һ��Ԫ��
__global__ void matAddKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	Cvalue = getElement(A, row, col) + getElement(B, row, col);
	setElement(C, row, col, Cvalue);
}
// �������kernel��2-D��ÿ���̼߳���һ��Ԫ��
__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
	int Cvalue = 0;
	int row = threadIdx.y + blockIdx.y * blockDim.y;//��ȡ���߳�������ľ����к�
	int col = threadIdx.x + blockIdx.x * blockDim.x;//��ȡ���߳�������ľ����к�

	for (int i = 0; i < A->width; ++i)//��ͨ�ľ���˷�
	{
		int a, b, c;
		a = A->elements[row*A->width + i];
		b = B->elements[i*B->width + col];
		c = a * b;
		Cvalue +=c;
	}
	setElement(C, row, col, Cvalue);//�ѽ��д�ص�C����
}
//using namespace std;
int main()
{
	int width = 1 << 2;
	int height = 1 << 2;
	Matrix *A, *B, *C, *D;
	

	int height_A = 16;
	int width_A = 4;
	int height_B = width_A;
	int width_B = 4;

	int width_result = width_B;
	int height_result = height_A;

	// �����й��ڴ�  ����ppt�ϵĿ�����ȥ������ ���൱���ǹ����ڴ��˰�
	hipMallocManaged((void**)&A, sizeof(Matrix));
	hipMallocManaged((void**)&B, sizeof(Matrix));
	hipMallocManaged((void**)&C, sizeof(Matrix));
	hipMallocManaged((void**)&D, sizeof(Matrix));

	hipMallocManaged((void**)&A->elements, height_A*width_A*sizeof(int));
	hipMallocManaged((void**)&B->elements, height_B*width_B * sizeof(int));
	hipMallocManaged((void**)&C->elements, height_result*width_result * sizeof(int));
	hipMallocManaged((void**)&D->elements, height_result*width_result * sizeof(int));

	A->height = height_A;
	A->width = width_A;
	B->height = height_B;
	B->width = width_B;
	C->height = height_result;
	C->width = width_result;
	D->height = height_result;
	D->width = width_result;

	for (int i = 0; i < height_A; i++)
	{
		for (int j = 0; j < width_A; j++)
		{
			A->elements[i*width_A + j]=rand()%10;
		}
	}
	for (int i = 0; i < height_B; i++)
	{
		for (int j = 0; j < width_B; j++)
		{
			B->elements[i*width_B + j]=rand()%10;
		}
	}
	dim3 blockSize(2,2);
	dim3 gridSize(width_B/blockSize.x,height_A/blockSize.y);
	// ִ��kernel
	matMulKernel <<< gridSize, blockSize >>> (A, B, C);
	 //ͬ��device ��֤�������ȷ����

	hipDeviceSynchronize();
	// ���ִ�н��


	std::cout << "A:" << std::endl;
	for (int i = 0; i < height_A; ++i)
	{
		for (int j = 0; j < width_A; j++)
		{
			printf("%3d ", A->elements[i*width_A + j]);
		}
		std::cout << std::endl;
	}
	std::cout << "B:" << std::endl;
	for (int i = 0; i < height_B; ++i)
	{
		for (int j = 0; j < width_B; j++)
		{
			printf("%3d ", B->elements[i*width_B + j]);
		}
		std::cout << std::endl;
	}
	std::cout << "\n\n\nA*B:" << std::endl;
	for (int i = 0; i < height_result; i++)
	{
		for (int j = 0; j < width_result; j++)
		{
			printf("%10d ", C->elements[i*width_result + j]);
		}
		std::cout << std::endl;
	}
	std::cout << std::endl;

	return 0;
}

/*
	1,ѧ����Nsight����
	2��ѧ��һЩ������cuda����
	3�����ʵ��2



*/
